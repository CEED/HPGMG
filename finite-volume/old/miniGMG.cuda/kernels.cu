#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------------------------------------------------------
// Samuel Williams
// SWWilliams@lbl.gov
// Lawrence Berkeley National Lab
//------------------------------------------------------------------------------------------------------------------------------
#include <stdio.h>
#include <stdint.h>
#include "defines.h"
#include "box.h"
#include "mg.h"

//------------------------------------------------------------------------------------------------------------------------------
#define enqueueEvent(id) domain->cudaEvents[domain->num_cudaEvents].level=level;domain->cudaEvents[domain->num_cudaEvents].type=id;hipEventRecord(domain->cudaEvents[domain->num_cudaEvents].event,0);domain->num_cudaEvents++;
//------------------------------------------------------------------------------------------------------------------------------
__constant__ int    faces[27] = {0,0,0,0,1,0,0,0,0,  0,1,0,1,0,1,0,1,0,  0,0,0,0,1,0,0,0,0};
//------------------------------------------------------------------------------------------------------------------------------


#warning  !!! Remember to try out __ldg()
#warning  !!! Everything is hard-coded for ghosts=1

#ifdef VL
#warning using vectorized smooth()
__global__ void __smooth_once_GSRB(subdomain_type * gpu_subdomains, int phi_id, int rhs_id, double a, double b, double h, int sweep, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // |--pencil--|--IJStride--|--pencil--|--??--|
  int IJStride = (VL - pencil - pencil) & ~0x0F;
  int ij = ((IJStride*blockIdx.x + pencil - pencil) & ~0xF) + threadIdx.x; // i.e. shift vector so that thread0 is 128-byte aligned
//int i = ij % pencil; // pencil is even and I will AND off all but the LSB
  int i = ij & 0x1; // pencil is even and I will AND off all but the LSB
  int j = ij / pencil;
  int k; 
  
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * lambda;
  if(threadIdx.x==0){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
    lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
  double *      lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane);
  #endif


  __shared__ double beta_i_ijk[VL];
  __shared__ double beta_j_ijk[VL];
  __shared__ double       temp[VL];
  double beta_k_ijk,beta_k_ijkp1;
  double    phi_ijk,   phi_ijkp1, phi_ijkm1;

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                                   int withinBounds = 1;
  if( threadIdx.x <  pencil           )withinBounds = 0;
  if( threadIdx.x >= pencil+IJStride  )withinBounds = 0;
  if( ij < pencil+1                   )withinBounds = 0;
  if( ij >=pencil*(subdomain_dim+1)-1 )withinBounds = 0;


  k = 0;
  int ijk = ij + k*plane;

     phi_ijkm1 =    phi[ijk-plane];
     phi_ijk   =    phi[ijk      ];
     phi_ijkp1 =    phi[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];

  int RedBlackUpdate = (i^j^sweep)&0x1;


  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
                                                                      temp[threadIdx.x] = phi_ijk;
    if( (threadIdx.x>=pencil)&&(threadIdx.x<pencil+IJStride+1) )beta_i_ijk[threadIdx.x] = beta_i[ijk];
    if( (threadIdx.x>=pencil)                                  )beta_j_ijk[threadIdx.x] = beta_j[ijk];
    __syncthreads(); // RAW guard

    if(withinBounds){
      double helmholtz_ijk = a*alpha[ijk]*phi_ijk - b*h2inv*(
        beta_i_ijk[threadIdx.x+     1] * ( temp[threadIdx.x+     1] - phi_ijk                  ) -
        beta_i_ijk[threadIdx.x       ] * ( phi_ijk                  - temp[threadIdx.x-     1] ) +
        beta_j_ijk[threadIdx.x+pencil] * ( temp[threadIdx.x+pencil] - phi_ijk                  ) -
        beta_j_ijk[threadIdx.x       ] * ( phi_ijk                  - temp[threadIdx.x-pencil] ) +
        beta_k_ijkp1                   * ( phi_ijkp1                - phi_ijk                  ) -
        beta_k_ijk                     * ( phi_ijk                  - phi_ijkm1                )
      );
      // GSRB
      double new_phi = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]);
      phi[ijk] = (RedBlackUpdate) ? new_phi : phi_ijk;
    }

    RedBlackUpdate=RedBlackUpdate^1;
    ijk+=plane;
    // rotate register pipeline...
       phi_ijkm1 =    phi_ijk;
       phi_ijk   =    phi_ijkp1;
       phi_ijkp1 =    phi[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // GSRB kernel

#else
#ifdef __LOCALITY_VIA_SHARED
//==============================================================================================================================================================
// shared+L1 version (relies on shared memory but you must still favorL1)
//==============================================================================================================================================================
__global__ void __smooth_once_GSRB(subdomain_type * gpu_subdomains, int phi_id, int rhs_id, double a, double b, double h, int sweep, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * lambda;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
    lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *      lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane+pencil+1);
  #endif


  __shared__ double beta_i_ijk[TBDIMY  ][TBDIMX+1];
  __shared__ double beta_j_ijk[TBDIMY+1][TBDIMX  ];
  __shared__ double       temp[TBDIMY+2][TBDIMX+2]; // always index with threadIdx.y+1,x+1
  double beta_k_ijk,beta_k_ijkp1;
  double    phi_ijk,   phi_ijkp1, phi_ijkm1;
  

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

     k = 0; 
     int ijk = k*plane + j*pencil + i;

     phi_ijkm1 = phi[ijk-plane];
     phi_ijk   = phi[ijk      ];
     phi_ijkp1 = phi[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];

  int RedBlackUpdate = (i^j^sweep) & 0x1;

  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
    // copy phi and beta_i/j to shared temp array...
                            temp[threadIdx.y+1][threadIdx.x+1] = phi_ijk;
    if(threadIdx.x==0)      temp[threadIdx.y+1][            0] = phi[ijk-1];
    if(threadIdx.y==0)      temp[            0][threadIdx.x+1] = phi[ijk-pencil];
    if(threadIdx.x==0)      temp[threadIdx.y+1][     TBDIMX+1] = phi[ijk+TBDIMX];
    if(threadIdx.y==0)      temp[     TBDIMY+1][threadIdx.x+1] = phi[ijk+TBDIMY*pencil];
                      beta_i_ijk[threadIdx.y  ][threadIdx.x  ] = beta_i[ijk];
    if(threadIdx.x==0)beta_i_ijk[threadIdx.y  ][     TBDIMX  ] = beta_i[ijk+TBDIMX];
                      beta_j_ijk[threadIdx.y  ][threadIdx.x  ] = beta_j[ijk];
    if(threadIdx.y==0)beta_j_ijk[     TBDIMY  ][threadIdx.x  ] = beta_j[ijk+TBDIMY*pencil];
    __syncthreads(); // RAW guard

    if(withinBounds){
      double helmholtz_ijk = a*alpha[ijk]*phi_ijk - b*h2inv*( 
        beta_i_ijk[threadIdx.y  ][threadIdx.x+1] * ( temp[threadIdx.y+1][threadIdx.x+2] - phi_ijk                            ) - 
        beta_i_ijk[threadIdx.y  ][threadIdx.x  ] * ( phi_ijk                            - temp[threadIdx.y+1][threadIdx.x  ] ) +
        beta_j_ijk[threadIdx.y+1][threadIdx.x  ] * ( temp[threadIdx.y+2][threadIdx.x+1] - phi_ijk                            ) -
        beta_j_ijk[threadIdx.y  ][threadIdx.x  ] * ( phi_ijk                            - temp[threadIdx.y  ][threadIdx.x+1] ) +
        beta_k_ijkp1                             * ( phi_ijkp1                          - phi_ijk                            ) -
        beta_k_ijk                               * ( phi_ijk                            - phi_ijkm1                          )
      );
      // GSRB
      double new_phi = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]);
      phi[ijk] = (RedBlackUpdate) ? new_phi : phi_ijk;
    }
 
    //if(withinBounds && ((RedBlackUpdate^k)&0x1)){ 
    //  phi[ijk] = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]); // GSRB
    //}
    //double new_phi = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]);
    //if(withinBounds){ 
    //  phi[ijk] = ((RedBlackUpdate^k)&0x1) ? new_phi : phi_ijk;
    //}

    RedBlackUpdate=RedBlackUpdate^1;
    ijk+=plane;
    // rotate register pipeline...
    phi_ijkm1    = phi_ijk;
    phi_ijk      = phi_ijkp1;
    phi_ijkp1    = phi[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // GSRB kernel
#else
//==============================================================================================================================================================
// cache version (relies solely on the L1/L2 cache hierarchy)
//==============================================================================================================================================================
__global__ void __smooth_once_GSRB(subdomain_type * gpu_subdomains, int phi_id, int rhs_id, double a, double b, double h, int sweep, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * lambda;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
    lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *      lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (plane+pencil+1);
  #endif


  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  int RedBlackUpdate = (i^j^sweep) & 0x1;

  for(k=0;k<subdomain_dim;k++){
    int ijk = k*plane + j*pencil + i;

    if(withinBounds){
      double phi_ijk = phi[ijk];
      double helmholtz_ijk = a*alpha[ijk]*phi[ijk] - b*h2inv*( 
        beta_i[ijk     +1] * ( phi[ijk     +1] - phi[ijk       ] ) -
        beta_i[ijk       ] * ( phi[ijk       ] - phi[ijk     -1] ) + 
        beta_j[ijk+pencil] * ( phi[ijk+pencil] - phi[ijk       ] ) -
        beta_j[ijk       ] * ( phi[ijk       ] - phi[ijk-pencil] ) + 
        beta_k[ijk +plane] * ( phi[ijk +plane] - phi[ijk       ] ) -
        beta_k[ijk       ] * ( phi[ijk       ] - phi[ijk -plane] )
      );
      // GSRB
      double new_phi = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]);
      phi[ijk] = (RedBlackUpdate) ? new_phi : phi_ijk;
    }
    //if(withinBounds && ((RedBlackUpdate^k)&0x1)){
    //  phi[ijk] = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]); // GSRB
    //}
    //double new_phi = phi_ijk - lambda[ijk]*(helmholtz_ijk - rhs[ijk]);
    //if(withinBounds){ 
    //  phi[ijk] = ((RedBlackUpdate^k)&0x1) ? new_phi : phi_ijk;
    //}
    RedBlackUpdate=RedBlackUpdate^1;
  } // for k
} // GSRB kernel
#endif
#endif // 1D vs 2D versions



//==================================================================================================================================================
__global__ void __dot(subdomain_type * gpu_subdomains, int id_a, int id_b, double *global_dot, int level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;


  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------
  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);
  double * grid_b = gpu_subdomains[box].levels[level].grids[id_b] + (plane+pencil+1);


  // register/shared pipelines ---------------------------------------------------------------------------------------------------------------------
  __shared__ double local_dot[TBDIMY][TBDIMX];
  local_dot[threadIdx.y][threadIdx.x] = 0.0;
  __syncthreads(); 

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  // sweep through k dimension ---------------------------------------------------------------------------------------------------------------------
  for(k=0;k<subdomain_dim;k++){ // reduction in z
    int ijk = k*plane + j*pencil + i;
    if(withinBounds){
      local_dot[threadIdx.y][threadIdx.x]+=grid_a[ijk]*grid_b[ijk]; // dot product
    }
  } // for k
  __syncthreads();  // ensure everyone has found the max for their column
  int y;
  for(y=1;y<TBDIMY;y++){ // reduction in y
    if(threadIdx.y==0)local_dot[0][threadIdx.x]+=local_dot[y][threadIdx.x];
  }
  __syncthreads();  // ensure all threads[0][x] have have their sum over y
  int x;
  for(x=1;x<TBDIMX;x++){ // reduction in x
    if((threadIdx.x==0)&&(threadIdx.y==0))local_dot[0][0]+=local_dot[0][x];
  }
  __syncthreads();  // ensure thread 0 has thread block's sum;

  // now try and do an atomic add with local_dot[0][0] and *global_dot ------------------------------------------------------------------------------
  if((threadIdx.x==0)&&(threadIdx.y==0)){
    double oldd,newd;
    unsigned long long oldull, newull, CASrv;
    oldd = *global_dot;
    oldull = __double_as_longlong(oldd);
    newd = oldd;
    newd+=local_dot[0][0];
    newull = __double_as_longlong(newd);
    while ((CASrv=atomicCAS((unsigned long long *)global_dot, oldull, newull)) != oldull){
      oldull = CASrv;
      newd = __longlong_as_double(oldull);
      newd+=local_dot[0][0];
      newull = __double_as_longlong(newd);
    }
    // when complete, host should hipMemcpy(&cpu_dot, gpu_dot, sizeof(double), hipMemcpyDeviceToHost); 
  }

} // dot product kernel


//==================================================================================================================================================
__global__ void __sum(subdomain_type * gpu_subdomains, int id_a, double *global_sum, int level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;


  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------
  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);


  // register/shared pipelines ---------------------------------------------------------------------------------------------------------------------
  __shared__ double local_sum[TBDIMY][TBDIMX];
  local_sum[threadIdx.y][threadIdx.x] = 0.0;
  __syncthreads(); 

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  // sweep through k dimension ---------------------------------------------------------------------------------------------------------------------
  for(k=0;k<subdomain_dim;k++){ // reduction in z
    int ijk = k*plane + j*pencil + i;
    if(withinBounds){
      local_sum[threadIdx.y][threadIdx.x]+=grid_a[ijk];
    }
  } // for k
  __syncthreads();  // ensure everyone has found the max for their column
  int y;
  for(y=1;y<TBDIMY;y++){ // reduction in y
    if(threadIdx.y==0)local_sum[0][threadIdx.x]+=local_sum[y][threadIdx.x];
  }
  __syncthreads();  // ensure all threads[0][x] have have their sum over y
  int x;
  for(x=1;x<TBDIMX;x++){ // reduction in x
    if((threadIdx.x==0)&&(threadIdx.y==0))local_sum[0][0]+=local_sum[0][x];
  }
  __syncthreads();  // ensure thread 0 has thread block's sum;

  // now try and do an atomic add with local_sum[0][0] and *global_sum ------------------------------------------------------------------------------
  if((threadIdx.x==0)&&(threadIdx.y==0)){
    double oldd,newd;
    unsigned long long oldull, newull, CASrv;
    oldd = *global_sum;
    oldull = __double_as_longlong(oldd);
    newd = oldd;
    newd+=local_sum[0][0];
    newull = __double_as_longlong(newd);
    while ((CASrv=atomicCAS((unsigned long long *)global_sum, oldull, newull)) != oldull){
      oldull = CASrv;
      newd = __longlong_as_double(oldull);
      newd+=local_sum[0][0];
      newull = __double_as_longlong(newd);
    }
    // when complete, host should hipMemcpy(&cpu_sum, gpu_sum, sizeof(double), hipMemcpyDeviceToHost); 
  }

} // sum product kernel


//==================================================================================================================================================
__global__ void __norm(subdomain_type * gpu_subdomains, int grid_id, double *global_norm, int level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;


  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------
  double *   grid = gpu_subdomains[box].levels[level].grids[grid_id] + (plane+pencil+1);


  // register/shared pipelines ---------------------------------------------------------------------------------------------------------------------
  __shared__ double max_norm[TBDIMY][TBDIMX];
  max_norm[threadIdx.y][threadIdx.x] = 0.0;
  __syncthreads(); 

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  // sweep through k dimension ---------------------------------------------------------------------------------------------------------------------
  for(k=0;k<subdomain_dim;k++){ // reduction in z
    int ijk = k*plane + j*pencil + i;
    double fabs_grid_ijk = fabs(grid[ijk]);
    if(withinBounds){
      if(fabs_grid_ijk>max_norm[threadIdx.y][threadIdx.x]){max_norm[threadIdx.y][threadIdx.x]=fabs_grid_ijk;} // max norm
    }
  } // for k
  __syncthreads();  // ensure everyone has found the max for their column
  int y;
  if(threadIdx.y==0)for(y=1;y<TBDIMY;y++){ // reduction in y
    if(max_norm[y][threadIdx.x]>max_norm[0][threadIdx.x]){max_norm[0][threadIdx.x]=max_norm[y][threadIdx.x];} // max norm
  }
  __syncthreads();  // ensure all threads in x have found the max for all their corresponding y's
  int x;
  if((threadIdx.x==0)&&(threadIdx.y==0))for(x=1;x<TBDIMX;x++){ // reduction in x
    if(max_norm[0][x]>max_norm[0][0]){max_norm[0][0]=max_norm[0][x];} // max norm
  }
  __syncthreads();  // ensure thread 0 has found the global max;

  // now try and do an atomic max with max_norm[0][0] and*global_norm ------------------------------------------------------------------------------
  if((threadIdx.x==0)&&(threadIdx.y==0)){
    double oldd,newd;
    unsigned long long oldull, newull, CASrv;
    oldd = *global_norm;
    oldull = __double_as_longlong(oldd);
    newd = oldd;if(max_norm[0][0]>newd)newd=max_norm[0][0];
    newull = __double_as_longlong(newd);
    while ((CASrv=atomicCAS((unsigned long long *)global_norm, oldull, newull)) != oldull){
      oldull = CASrv;
      newd = __longlong_as_double(oldull);
      if(max_norm[0][0]>newd)newd=max_norm[0][0];
      newull = __double_as_longlong(newd);
    }
    // when complete, host should hipMemcpy(&cpu_norm, gpu_norm, sizeof(double), hipMemcpyDeviceToHost); 
  }

} // norm kernel



//==================================================================================================================================================
#ifdef VL
__global__ void __residual(subdomain_type * gpu_subdomains, int res_id, int phi_id, int rhs_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // |--pencil--|--IJStride--|--pencil--|--??--|
  int IJStride = (VL - pencil - pencil) & ~0x0F;
  int ij = ((IJStride*blockIdx.x + pencil - pencil) & ~0xF) + threadIdx.x; // i.e. shift vector so that thread0 is 128-byte aligned
  int k; 
 
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * res;
  if(threadIdx.x==0){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
       res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
  double *         res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane);
  #endif

 
  __shared__ double beta_i_ijk[VL];
  __shared__ double beta_j_ijk[VL];
  __shared__ double       temp[VL];
  double beta_k_ijk,beta_k_ijkp1;
  double    phi_ijk,   phi_ijkp1, phi_ijkm1;

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                                   int withinBounds = 1;
  if( threadIdx.x <  pencil           )withinBounds = 0;
  if( threadIdx.x >= pencil+IJStride  )withinBounds = 0;
  if( ij < pencil+1                   )withinBounds = 0;
  if( ij >=pencil*(subdomain_dim+1)-1 )withinBounds = 0;

  k = 0;
  int ijk = ij + k*plane;

     phi_ijkm1 =    phi[ijk-plane];
     phi_ijk   =    phi[ijk      ];
     phi_ijkp1 =    phi[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];


  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
                                                                      temp[threadIdx.x] = phi_ijk;
    if( (threadIdx.x>=pencil)&&(threadIdx.x<pencil+IJStride+1) )beta_i_ijk[threadIdx.x] = beta_i[ijk];
    if( (threadIdx.x>=pencil)                                  )beta_j_ijk[threadIdx.x] = beta_j[ijk];
    __syncthreads(); // RAW guard

    double helmholtz_ijk;
    if(withinBounds)
    helmholtz_ijk = a*alpha[ijk]*phi_ijk - b*h2inv*(
      beta_i_ijk[threadIdx.x+     1] * ( temp[threadIdx.x+     1] - phi_ijk                  ) -
      beta_i_ijk[threadIdx.x       ] * ( phi_ijk                  - temp[threadIdx.x-     1] ) +
      beta_j_ijk[threadIdx.x+pencil] * ( temp[threadIdx.x+pencil] - phi_ijk                  ) -
      beta_j_ijk[threadIdx.x       ] * ( phi_ijk                  - temp[threadIdx.x-pencil] ) +
      beta_k_ijkp1                   * ( phi_ijkp1                - phi_ijk                  ) -
      beta_k_ijk                     * ( phi_ijk                  - phi_ijkm1                )
    );

    if(withinBounds)res[ijk] = rhs[ijk] - helmholtz_ijk;

    ijk+=plane;
    // rotate register pipeline...
    phi_ijkm1    = phi_ijk;
    phi_ijk      = phi_ijkp1;
    phi_ijkp1    = phi[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    if(withinBounds)
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // residual kernel

#else
#ifdef __LOCALITY_VIA_SHARED
//==============================================================================================================================================================
// shared+L1 version (relies on shared memory but you must still favorL1)
//==============================================================================================================================================================
__global__ void __residual(subdomain_type * gpu_subdomains, int res_id, int phi_id, int rhs_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * res;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
       res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *         res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane+pencil+1);
  #endif


  __shared__ double beta_i_ijk[TBDIMY  ][TBDIMX+1];
  __shared__ double beta_j_ijk[TBDIMY+1][TBDIMX  ];
  __shared__ double       temp[TBDIMY+2][TBDIMX+2]; // always index with threadIdx.y+1,x+1
  double beta_k_ijk,beta_k_ijkp1;
  double    phi_ijk,   phi_ijkp1, phi_ijkm1;
  

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

     k = 0; 
     int ijk = k*plane + j*pencil + i;

     phi_ijkm1 = phi[ijk-plane];
     phi_ijk   = phi[ijk      ];
     phi_ijkp1 = phi[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];

  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
    // copy phi and beta_i/j to shared temp array...
                            temp[threadIdx.y+1][threadIdx.x+1] = phi_ijk;
    if(threadIdx.x==0)      temp[threadIdx.y+1][            0] = phi[ijk-1];
    if(threadIdx.y==0)      temp[            0][threadIdx.x+1] = phi[ijk-pencil];
    if(threadIdx.x==0)      temp[threadIdx.y+1][     TBDIMX+1] = phi[ijk+TBDIMX];
    if(threadIdx.y==0)      temp[     TBDIMY+1][threadIdx.x+1] = phi[ijk+TBDIMY*pencil];
                      beta_i_ijk[threadIdx.y  ][threadIdx.x  ] = beta_i[ijk];
    if(threadIdx.x==0)beta_i_ijk[threadIdx.y  ][     TBDIMX  ] = beta_i[ijk+TBDIMX];
                      beta_j_ijk[threadIdx.y  ][threadIdx.x  ] = beta_j[ijk];
    if(threadIdx.y==0)beta_j_ijk[     TBDIMY  ][threadIdx.x  ] = beta_j[ijk+TBDIMY*pencil];
    __syncthreads(); // RAW guard

    double helmholtz_ijk = a*alpha[ijk]*phi_ijk - b*h2inv*( 
      beta_i_ijk[threadIdx.y  ][threadIdx.x+1] * ( temp[threadIdx.y+1][threadIdx.x+2] - phi_ijk                            ) - 
      beta_i_ijk[threadIdx.y  ][threadIdx.x  ] * ( phi_ijk                            - temp[threadIdx.y+1][threadIdx.x  ] ) +
      beta_j_ijk[threadIdx.y+1][threadIdx.x  ] * ( temp[threadIdx.y+2][threadIdx.x+1] - phi_ijk                            ) -
      beta_j_ijk[threadIdx.y  ][threadIdx.x  ] * ( phi_ijk                            - temp[threadIdx.y  ][threadIdx.x+1] ) +
      beta_k_ijkp1                             * ( phi_ijkp1                          - phi_ijk                            ) -
      beta_k_ijk                               * ( phi_ijk                            - phi_ijkm1                          )
    );
 
    if(withinBounds)res[ijk] = rhs[ijk] - helmholtz_ijk;

    ijk+=plane;
    // rotate register pipeline...
    phi_ijkm1    = phi_ijk;
    phi_ijk      = phi_ijkp1;
    phi_ijkp1    = phi[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // residual kernel
#else
//==============================================================================================================================================================
// cache version (relies solely on the L1/L2 cache hierarchy)
//==============================================================================================================================================================
__global__ void __residual(subdomain_type * gpu_subdomains, int res_id, int phi_id, int rhs_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;


  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * phi;
  __shared__ double * rhs;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double * res;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
       phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
       rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
       res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *         phi = gpu_subdomains[box].levels[level].grids[  phi_id] + (plane+pencil+1);
  double *         rhs = gpu_subdomains[box].levels[level].grids[  rhs_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *         res = gpu_subdomains[box].levels[level].grids[  res_id] + (plane+pencil+1);
  #endif



  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  for(k=0;k<subdomain_dim;k++){
    int ijk = k*plane + j*pencil + i;
    double helmholtz_ijk = a*alpha[ijk]*phi[ijk] - b*h2inv*(
      beta_i[ijk     +1] * ( phi[ijk     +1] - phi[ijk       ] ) -
      beta_i[ijk       ] * ( phi[ijk       ] - phi[ijk     -1] ) +
      beta_j[ijk+pencil] * ( phi[ijk+pencil] - phi[ijk       ] ) -
      beta_j[ijk       ] * ( phi[ijk       ] - phi[ijk-pencil] ) +
      beta_k[ijk +plane] * ( phi[ijk +plane] - phi[ijk       ] ) -
      beta_k[ijk       ] * ( phi[ijk       ] - phi[ijk -plane] )
    );
    if(withinBounds)res[ijk] = rhs[ijk] - helmholtz_ijk;
  } // for k
} // residual kernel
#endif
#endif // VL

//=============================================================================================================================================================
#ifdef VL
__global__ void __apply_op(subdomain_type * gpu_subdomains, int  Ax_id, int x_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // |--pencil--|--IJStride--|--pencil--|--??--|
  int IJStride = (VL - pencil - pencil) & ~0x0F;
  int ij = ((IJStride*blockIdx.x + pencil - pencil) & ~0xF) + threadIdx.x; // i.e. shift vector so that thread0 is 128-byte aligned
  int k; 
 
  #ifdef __POINTERS_IN_SHARED
  __shared__ double * x;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double *  Ax;
  if(threadIdx.x==0){
         x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
        Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane);
  }
  __syncthreads();
  #else
  double *           x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane);
  double *          Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane);
  #endif

 
  __shared__ double beta_i_ijk[VL];
  __shared__ double beta_j_ijk[VL];
  __shared__ double       temp[VL];
  double beta_k_ijk,beta_k_ijkp1;
  double      x_ijk,     x_ijkp1,   x_ijkm1;

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                                   int withinBounds = 1;
  if( threadIdx.x <  pencil           )withinBounds = 0;
  if( threadIdx.x >= pencil+IJStride  )withinBounds = 0;
  if( ij < pencil+1                   )withinBounds = 0;
  if( ij >=pencil*(subdomain_dim+1)-1 )withinBounds = 0;

  k = 0;
  int ijk = ij + k*plane;

       x_ijkm1 =      x[ijk-plane];
       x_ijk   =      x[ijk      ];
       x_ijkp1 =      x[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];


  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
                                                                      temp[threadIdx.x] =   x_ijk;
    if( (threadIdx.x>=pencil)&&(threadIdx.x<pencil+IJStride+1) )beta_i_ijk[threadIdx.x] = beta_i[ijk];
    if( (threadIdx.x>=pencil)                                  )beta_j_ijk[threadIdx.x] = beta_j[ijk];
    __syncthreads(); // RAW guard

    double helmholtz_ijk;
    if(withinBounds)
    helmholtz_ijk = a*alpha[ijk]*  x_ijk - b*h2inv*(
      beta_i_ijk[threadIdx.x+     1] * ( temp[threadIdx.x+     1] -   x_ijk                  ) -
      beta_i_ijk[threadIdx.x       ] * (   x_ijk                  - temp[threadIdx.x-     1] ) +
      beta_j_ijk[threadIdx.x+pencil] * ( temp[threadIdx.x+pencil] -   x_ijk                  ) -
      beta_j_ijk[threadIdx.x       ] * (   x_ijk                  - temp[threadIdx.x-pencil] ) +
      beta_k_ijkp1                   * (   x_ijkp1                -   x_ijk                  ) -
      beta_k_ijk                     * (   x_ijk                  -   x_ijkm1                )
    );

    if(withinBounds) Ax[ijk] = helmholtz_ijk;

    ijk+=plane;
    // rotate register pipeline...
      x_ijkm1    =   x_ijk;
      x_ijk      =   x_ijkp1;
      x_ijkp1    =   x[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    if(withinBounds)
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // residual kernel

#else
#ifdef __LOCALITY_VIA_SHARED
//==============================================================================================================================================================
// shared+L1 version (relies on shared memory but you must still favorL1)
//==============================================================================================================================================================
__global__ void __apply_op(subdomain_type * gpu_subdomains, int  Ax_id, int   x_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double *   x;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double *  Ax;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
         x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
        Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *           x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *          Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane+pencil+1);
  #endif


  __shared__ double beta_i_ijk[TBDIMY  ][TBDIMX+1];
  __shared__ double beta_j_ijk[TBDIMY+1][TBDIMX  ];
  __shared__ double       temp[TBDIMY+2][TBDIMX+2]; // always index with threadIdx.y+1,x+1
  double beta_k_ijk,beta_k_ijkp1;
  double      x_ijk,     x_ijkp1,   x_ijkm1;
  

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

     k = 0; 
     int ijk = k*plane + j*pencil + i;

       x_ijkm1 =   x[ijk-plane];
       x_ijk   =   x[ijk      ];
       x_ijkp1 =   x[ijk+plane];
  beta_k_ijk   = beta_k[ijk      ];
  beta_k_ijkp1 = beta_k[ijk+plane];

  for(k=0;k<subdomain_dim;k++){

    __syncthreads(); // WAR guard
    // copy   x and beta_i/j to shared temp array...
                            temp[threadIdx.y+1][threadIdx.x+1] =   x_ijk;
    if(threadIdx.x==0)      temp[threadIdx.y+1][            0] =   x[ijk-1];
    if(threadIdx.y==0)      temp[            0][threadIdx.x+1] =   x[ijk-pencil];
    if(threadIdx.x==0)      temp[threadIdx.y+1][     TBDIMX+1] =   x[ijk+TBDIMX];
    if(threadIdx.y==0)      temp[     TBDIMY+1][threadIdx.x+1] =   x[ijk+TBDIMY*pencil];
                      beta_i_ijk[threadIdx.y  ][threadIdx.x  ] = beta_i[ijk];
    if(threadIdx.x==0)beta_i_ijk[threadIdx.y  ][     TBDIMX  ] = beta_i[ijk+TBDIMX];
                      beta_j_ijk[threadIdx.y  ][threadIdx.x  ] = beta_j[ijk];
    if(threadIdx.y==0)beta_j_ijk[     TBDIMY  ][threadIdx.x  ] = beta_j[ijk+TBDIMY*pencil];
    __syncthreads(); // RAW guard

    double helmholtz_ijk = a*alpha[ijk]*  x_ijk - b*h2inv*( 
      beta_i_ijk[threadIdx.y  ][threadIdx.x+1] * ( temp[threadIdx.y+1][threadIdx.x+2] -   x_ijk                            ) - 
      beta_i_ijk[threadIdx.y  ][threadIdx.x  ] * (   x_ijk                            - temp[threadIdx.y+1][threadIdx.x  ] ) +
      beta_j_ijk[threadIdx.y+1][threadIdx.x  ] * ( temp[threadIdx.y+2][threadIdx.x+1] -   x_ijk                            ) -
      beta_j_ijk[threadIdx.y  ][threadIdx.x  ] * (   x_ijk                            - temp[threadIdx.y  ][threadIdx.x+1] ) +
      beta_k_ijkp1                             * (   x_ijkp1                          -   x_ijk                            ) -
      beta_k_ijk                               * (   x_ijk                            -   x_ijkm1                          )
    );
 
    if(withinBounds) Ax[ijk] = helmholtz_ijk;

    ijk+=plane;
    // rotate register pipeline...
      x_ijkm1    =   x_ijk;
      x_ijk      =   x_ijkp1;
      x_ijkp1    =   x[ijk+plane];
    beta_k_ijk   = beta_k_ijkp1;
    beta_k_ijkp1 = beta_k[ijk+plane];

  } // for k
} // residual kernel
#else
//==============================================================================================================================================================
// cache version (relies solely on the L1/L2 cache hierarchy)
//==============================================================================================================================================================
__global__ void __apply_op(subdomain_type * gpu_subdomains, int  Ax_id, int   x_id, double a, double b, double h, int level){

  double h2inv = 1.0/(h*h);
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;


  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  #ifdef __POINTERS_IN_SHARED
  __shared__ double *   x;
  __shared__ double * alpha;
  __shared__ double * beta_i;
  __shared__ double * beta_j;
  __shared__ double * beta_k;
  __shared__ double *  Ax;
  if((threadIdx.x==0)&&(threadIdx.y==0)){
         x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane+pencil+1);
     alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
    beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
    beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
    beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
        Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane+pencil+1);
  }
  __syncthreads();
  #else
  double *           x = gpu_subdomains[box].levels[level].grids[    x_id] + (plane+pencil+1);
  double *       alpha = gpu_subdomains[box].levels[level].grids[ __alpha] + (plane+pencil+1);
  double *      beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (plane+pencil+1);
  double *      beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (plane+pencil+1);
  double *      beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (plane+pencil+1);
  double *          Ax = gpu_subdomains[box].levels[level].grids[   Ax_id] + (plane+pencil+1);
  #endif



  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  for(k=0;k<subdomain_dim;k++){
    int ijk = k*plane + j*pencil + i;
    double helmholtz_ijk = a*alpha[ijk]*  x[ijk] - b*h2inv*(
      beta_i[ijk     +1] * (   x[ijk     +1] -   x[ijk       ] ) -
      beta_i[ijk       ] * (   x[ijk       ] -   x[ijk     -1] ) +
      beta_j[ijk+pencil] * (   x[ijk+pencil] -   x[ijk       ] ) -
      beta_j[ijk       ] * (   x[ijk       ] -   x[ijk-pencil] ) +
      beta_k[ijk +plane] * (   x[ijk +plane] -   x[ijk       ] ) -
      beta_k[ijk       ] * (   x[ijk       ] -   x[ijk -plane] )
    );
    if(withinBounds) Ax[ijk] = helmholtz_ijk;
  } // for k
} // residual kernel
#endif
#endif // VL

//=============================================================================================================================================================
__global__ void __restriction_betas(subdomain_type * gpu_subdomains, int fine_level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int   fine_dim    = gpu_subdomains[box].levels[fine_level  ].dim.i;
  int   fine_pencil = gpu_subdomains[box].levels[fine_level  ].pencil;
  int   fine_plane  = gpu_subdomains[box].levels[fine_level  ].plane;
  int coarse_dim    = gpu_subdomains[box].levels[fine_level+1].dim.i;
  int coarse_pencil = gpu_subdomains[box].levels[fine_level+1].pencil;
  int coarse_plane  = gpu_subdomains[box].levels[fine_level+1].plane;

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                    int withinCoarseBounds = 1;
  if((i>>1)>=coarse_dim)withinCoarseBounds = 0;
  if((j>>1)>=coarse_dim)withinCoarseBounds = 0;

  // restrict beta_i  (== face in jk)
  double * beta_f = gpu_subdomains[box].levels[fine_level  ].grids[__beta_i] + (  fine_plane+  fine_pencil+1);
  double * beta_c = gpu_subdomains[box].levels[fine_level+1].grids[__beta_i] + (coarse_plane+coarse_pencil+1);
  for(k=0;k<fine_dim;k+=2){
    int   fine_ijk = (k   )*  fine_plane + (j   )*  fine_pencil + (i   );
    int coarse_ijk = (k>>1)*coarse_plane + (j>>1)*coarse_pencil + (i>>1);
    if(withinCoarseBounds){
    if(!((threadIdx.x|threadIdx.y)&0x1)){ // i.e. x and y LSB are 0
      beta_c[coarse_ijk] = ( beta_f[fine_ijk            ]+beta_f[fine_ijk+fine_pencil           ] +
                             beta_f[fine_ijk+fine_plane ]+beta_f[fine_ijk+fine_pencil+fine_plane] ) * 0.25;
    }}
  } // for k
  // restrict beta_j  (== face in ik)
  beta_f = gpu_subdomains[box].levels[fine_level  ].grids[__beta_j] + (  fine_plane+  fine_pencil+1);
  beta_c = gpu_subdomains[box].levels[fine_level+1].grids[__beta_j] + (coarse_plane+coarse_pencil+1);
  for(k=0;k<fine_dim;k+=2){
    int   fine_ijk = (k   )*  fine_plane + (j   )*  fine_pencil + (i   );
    int coarse_ijk = (k>>1)*coarse_plane + (j>>1)*coarse_pencil + (i>>1);
    if(withinCoarseBounds){
    if(!((threadIdx.x|threadIdx.y)&0x1)){ // i.e. x and y LSB are 0
      beta_c[coarse_ijk] = ( beta_f[fine_ijk            ]+beta_f[fine_ijk+1                     ] +
                             beta_f[fine_ijk+fine_plane ]+beta_f[fine_ijk+1+fine_plane          ] ) * 0.25;
    }}
  } // for k
  // restrict beta_k  (== face in ij)
  beta_f = gpu_subdomains[box].levels[fine_level  ].grids[__beta_k] + (  fine_plane+  fine_pencil+1);
  beta_c = gpu_subdomains[box].levels[fine_level+1].grids[__beta_k] + (coarse_plane+coarse_pencil+1);
  for(k=0;k<fine_dim;k+=2){
    int   fine_ijk = (k   )*  fine_plane + (j   )*  fine_pencil + (i   );
    int coarse_ijk = (k>>1)*coarse_plane + (j>>1)*coarse_pencil + (i>>1);
    if(withinCoarseBounds){
    if(!((threadIdx.x|threadIdx.y)&0x1)){ // i.e. x and y LSB are 0
      beta_c[coarse_ijk] = ( beta_f[fine_ijk            ]+beta_f[fine_ijk+1                     ] +
                             beta_f[fine_ijk+fine_pencil]+beta_f[fine_ijk+1+fine_pencil         ] ) * 0.25;
    }}
  } // for k
} // restriction of face-centered coefficients kernel

//=============================================================================================================================================================
__global__ void __restriction(subdomain_type * gpu_subdomains, int fine_id, int coarse_id, int fine_level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int   fine_dim    = gpu_subdomains[box].levels[fine_level  ].dim.i;
  int   fine_pencil = gpu_subdomains[box].levels[fine_level  ].pencil;
  int   fine_plane  = gpu_subdomains[box].levels[fine_level  ].plane;
  int coarse_dim    = gpu_subdomains[box].levels[fine_level+1].dim.i;
  int coarse_pencil = gpu_subdomains[box].levels[fine_level+1].pencil;
  int coarse_plane  = gpu_subdomains[box].levels[fine_level+1].plane;

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                    int withinCoarseBounds = 1;
  if((i>>1)>=coarse_dim)withinCoarseBounds = 0;
  if((j>>1)>=coarse_dim)withinCoarseBounds = 0;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  double *   fine = gpu_subdomains[box].levels[fine_level  ].grids[  fine_id] + (  fine_plane+  fine_pencil+1);
  double * coarse = gpu_subdomains[box].levels[fine_level+1].grids[coarse_id] + (coarse_plane+coarse_pencil+1);

  // FIX - make coarse-oriented, not fine oriented...
  for(k=0;k<fine_dim;k+=2){
    int   fine_ijk = (k   )*  fine_plane + (j   )*  fine_pencil + (i   );
    int coarse_ijk = (k>>1)*coarse_plane + (j>>1)*coarse_pencil + (i>>1);

    if(withinCoarseBounds){
    if(!((threadIdx.x|threadIdx.y)&0x1)){ // i.e. x and y LSB are 0
      coarse[coarse_ijk] = 0.125 * (
                                   fine[fine_ijk                         ] +
                                   fine[fine_ijk                       +1] +
                                   fine[fine_ijk           +fine_pencil  ] +
                                   fine[fine_ijk           +fine_pencil+1] +
                                   fine[fine_ijk+fine_plane              ] +
                                   fine[fine_ijk+fine_plane            +1] +
                                   fine[fine_ijk+fine_plane+fine_pencil  ] +
                                   fine[fine_ijk+fine_plane+fine_pencil+1]
                                   );
    }}
  } // for k
} // restriction kernel

//=============================================================================================================================================================
__global__ void __interpolation(subdomain_type * gpu_subdomains, int coarse_id, int fine_id, int fine_level){

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int   fine_dim    = gpu_subdomains[box].levels[fine_level  ].dim.i;
  int   fine_pencil = gpu_subdomains[box].levels[fine_level  ].pencil;
  int   fine_plane  = gpu_subdomains[box].levels[fine_level  ].plane;
  int coarse_pencil = gpu_subdomains[box].levels[fine_level+1].pencil;
  int coarse_plane  = gpu_subdomains[box].levels[fine_level+1].plane;

  // construct pointers to element (0,0,0) in each array for the current subdomain -----------------------------------------------------------------------------
  double *   fine = gpu_subdomains[box].levels[fine_level  ].grids[  fine_id] + (  fine_plane+  fine_pencil+1);
  double * coarse = gpu_subdomains[box].levels[fine_level+1].grids[coarse_id] + (coarse_plane+coarse_pencil+1);

  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                    int   withinFineBounds = 1;
  if( i    >=  fine_dim)  withinFineBounds = 0;
  if( j    >=  fine_dim)  withinFineBounds = 0;
//                  int withinCoarseBounds = 1;
//if((i>>1)>=coarse_dim)withinCoarseBounds = 0;
//if((j>>1)>=coarse_dim)withinCoarseBounds = 0;

  for(k=0;k<fine_dim;k+=2){
    int   fine_ijk = (k   )*  fine_plane + (j   )*  fine_pencil + (i   );
    int coarse_ijk = (k>>1)*coarse_plane + (j>>1)*coarse_pencil + (i>>1);

    if(withinFineBounds){
      fine[fine_ijk           ] += coarse[coarse_ijk];
      fine[fine_ijk+fine_plane] += coarse[coarse_ijk];
    }
   
  } // for k
} // interpolation kernel

//==============================================================================================================================================================
__global__ void __grid_to_surface_buffers(subdomain_type * gpu_subdomains, int grid_id, int level){

  // ASSUME: ThreadBlock=(Dim x 1 x 1), Grid=(1,Dim,subdomains)

  //  |\
  //  |  \
  //  |\   \
  //  |  \   \
  //  |\   \  |   
  //  |  \   \|   ^ blockIdx.y (blockIdx.x is unused for grid dims<512)
  //  |\   \  |   |
  //  |  \   \|   |
  //   \   \  |   |
  //     \   \|
  //       \  |
  //         \|
  //   ---->
  //  threadIdx.x (threadIdx.y is unused)

  // FIX, what if ghosts > 1 ???

  int box = blockIdx.z;  // CUDA 4 !!!


  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // do the 6 faces...
  int di,dj,dk;
  for(dk=-1;dk<=1;dk++){
  for(dj=-1;dj<=1;dj++){
  for(di=-1;di<=1;di++){int n=13+di+3*dj+9*dk;if(faces[n]){
    int low_i,low_j,low_k;
    int elementStride;
    int  pencilStride;
    switch(di){
      case -1:low_i=  1;break;
      case  0:low_i=  1;break;
      case  1:low_i=subdomain_dim;break;
    };
    switch(dj){
      case -1:low_j=  1;break;
      case  0:low_j=  1;break;
      case  1:low_j=subdomain_dim;break;
    };
    switch(dk){
      case -1:low_k=  1;break;
      case  0:low_k=  1;break;
      case  1:low_k=subdomain_dim;break;
    };
    if(di==0)elementStride =      1; // ij and ik faces
        else elementStride = pencil; // jk faces
    if(dk==0) pencilStride =  plane; // ik and jk faces
        else  pencilStride = pencil; // ij faces
    int ijk = low_k*plane + low_j*pencil + low_i + pencilStride*blockIdx.y + elementStride*threadIdx.x;
    int b = subdomain_dim*blockIdx.y + threadIdx.x;

    double * surface_buf = gpu_subdomains[box].levels[level].surface_bufs[n];
    double * grid        = gpu_subdomains[box].levels[level].grids[grid_id];
 
    #ifdef __USE_LDG
    surface_buf[b] = __ldg(grid+ijk);
    #else
    surface_buf[b] = grid[ijk];
    #endif
  }}}}
}
//==============================================================================================================================================================
__global__ void __ghost_buffers_to_grid(subdomain_type * gpu_subdomains, int grid_id, int level){


  int box = blockIdx.z;  // CUDA 4 !!!


  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  // do the 6 faces...
  int di,dj,dk;
  for(dk=-1;dk<=1;dk++){
  for(dj=-1;dj<=1;dj++){
  for(di=-1;di<=1;di++){int n=13+di+3*dj+9*dk;if(faces[n]){
    int low_i,low_j,low_k;
    int elementStride;
    int  pencilStride;
    switch(di){
      case -1:low_i=              0;break;
      case  0:low_i=              1;break;
      case  1:low_i=subdomain_dim+1;break;
    };
    switch(dj){
      case -1:low_j=              0;break;
      case  0:low_j=              1;break;
      case  1:low_j=subdomain_dim+1;break;
    };
    switch(dk){
      case -1:low_k=              0;break;
      case  0:low_k=              1;break;
      case  1:low_k=subdomain_dim+1;break;
    };
    if(di==0)elementStride =      1; // ij and ik faces
        else elementStride = pencil; // jk faces
    if(dk==0) pencilStride =  plane; // ik and jk faces
        else  pencilStride = pencil; // ij faces
    int ijk = low_k*plane + low_j*pencil + low_i + pencilStride*blockIdx.y + elementStride*threadIdx.x;
    int b = subdomain_dim*blockIdx.y + threadIdx.x;

    double * ghost_buf = gpu_subdomains[box].levels[level].ghost_bufs[n];
    double * grid      = gpu_subdomains[box].levels[level].grids[grid_id];
 
    grid[ijk] = ghost_buf[b];
  }}}}
}


//==============================================================================================================================================================
__global__ void __surface_buffers_to_ghost_buffers(subdomain_type * gpu_subdomains, int grid_id, int level, int MyRank){
  int recvBox = blockIdx.z;  // CUDA 4 !!!
  int subdomain_dim = gpu_subdomains[recvBox].levels[level].dim.i;
  int n;
  for(n=0;n<27;n++)
  if( faces[n] && (gpu_subdomains[recvBox].neighbors[n].rank == MyRank) ){
    int sendBox = gpu_subdomains[recvBox].neighbors[n].local_index;
    double *   ghost_buf = gpu_subdomains[recvBox].levels[level].ghost_bufs[n];
    double * surface_buf = gpu_subdomains[sendBox].levels[level].surface_bufs[26-n];
    int b = subdomain_dim*blockIdx.y + threadIdx.x;
    ghost_buf[b] = surface_buf[b];
  }
}

//==============================================================================================================================================================
#ifdef __MPI
__global__ void __surface_buffers_to_send_buffer(subdomain_type * gpu_subdomains, double ** gpu_send_buffer, double ** gpu_recv_buffer, int grid_id, int level, int MyRank){
  int sendBox = blockIdx.z;  // CUDA 4 !!!
  int subdomain_dim = gpu_subdomains[sendBox].levels[level].dim.i;
  int n;
  int FaceSizeAtLevel = subdomain_dim*subdomain_dim;
  for(n=0;n<27;n++)
  if( faces[n] &&   (gpu_subdomains[sendBox].neighbors[n].rank != MyRank) ){
    int        buf = gpu_subdomains[sendBox].neighbors[n].send.buf;
    int FaceOffset = gpu_subdomains[sendBox].neighbors[n].send.offset.faces;
    double * surface_buf = gpu_subdomains[sendBox].levels[level].surface_bufs[n];
    double *    send_buf = gpu_send_buffer[buf] + FaceSizeAtLevel*FaceOffset;
    int b = subdomain_dim*blockIdx.y + threadIdx.x;
    send_buf[b] = surface_buf[b];
  }
}

__global__ void __recv_buffer_to_ghost_buffers(subdomain_type * gpu_subdomains, double ** gpu_send_buffer, double ** gpu_recv_buffer, int grid_id, int level, int MyRank){
  int recvBox = blockIdx.z;  // CUDA 4 !!!
  int subdomain_dim = gpu_subdomains[recvBox].levels[level].dim.i;
  int n;
  int FaceSizeAtLevel = subdomain_dim*subdomain_dim;
  for(n=0;n<27;n++)
  if( faces[n] &&   (gpu_subdomains[recvBox].neighbors[n].rank != MyRank) ){
    int        buf = gpu_subdomains[recvBox].neighbors[n].recv.buf;
    int FaceOffset = gpu_subdomains[recvBox].neighbors[n].recv.offset.faces;
    double *   ghost_buf = gpu_subdomains[recvBox].levels[level].ghost_bufs[n];
    double *    recv_buf = gpu_recv_buffer[buf] + FaceSizeAtLevel*FaceOffset;
    int b = subdomain_dim*blockIdx.y + threadIdx.x;
    ghost_buf[b] = recv_buf[b];
  }
}
#endif

 
//==============================================================================================================================================================
__global__ void __initialize_grid_to_scalar(subdomain_type * gpu_subdomains, int grid_id, double h, double value, int level){
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid = gpu_subdomains[box].levels[level].grids[grid_id] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds)grid[ijk] = value;
    ijk+=plane;
  } // for k
} // iniitalize kernel


//==============================================================================================================================================================
__global__ void __scale_grid(subdomain_type * gpu_subdomains, int id_c, double scale_a, int id_a, int level){ // c=scale_a*id_a
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);
  double * grid_c = gpu_subdomains[box].levels[level].grids[id_c] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds)grid_c[ijk] = scale_a*grid_a[ijk];
    ijk+=plane;
  } // for k
} // scale_grid kernel


//==============================================================================================================================================================
__global__ void __mul_grids(subdomain_type * gpu_subdomains, int id_c, double scale, int id_a, int id_b, int level){ // c=scale*id_a*id_b
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);
  double * grid_b = gpu_subdomains[box].levels[level].grids[id_b] + (plane+pencil+1);
  double * grid_c = gpu_subdomains[box].levels[level].grids[id_c] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds)grid_c[ijk] = scale*grid_a[ijk]*grid_b[ijk];
    ijk+=plane;
  } // for k
} // mul_grids kernel


//==============================================================================================================================================================
__global__ void __shift_grid(subdomain_type * gpu_subdomains, int id_c, int id_a, double shift_a, int level){ // id_c=id_a + shift_a
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid_c = gpu_subdomains[box].levels[level].grids[id_c] + (plane+pencil+1);
  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds)grid_c[ijk] = grid_a[ijk] + shift_a;
    ijk+=plane;
  } // for k
} // shift_grid kernel


//==============================================================================================================================================================
__global__ void __add_grids(subdomain_type * gpu_subdomains, int id_c, double scale_a, int id_a, double scale_b, int id_b, int level){ // c=scale_a*id_a + scale_b*id_b
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid_a = gpu_subdomains[box].levels[level].grids[id_a] + (plane+pencil+1);
  double * grid_b = gpu_subdomains[box].levels[level].grids[id_b] + (plane+pencil+1);
  double * grid_c = gpu_subdomains[box].levels[level].grids[id_c] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds)grid_c[ijk] = scale_a*grid_a[ijk] + scale_b*grid_b[ijk];
    ijk+=plane;
  } // for k
} // add_grids kernel


//==============================================================================================================================================================
__global__ void __rebuild_lambda(subdomain_type * gpu_subdomains, double a, double b, double h, int level){ 
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double h2inv = 1.0/(h*h);
  double * alpha  = gpu_subdomains[box].levels[level].grids[__alpha ] + (1+pencil+plane);
  double * beta_i = gpu_subdomains[box].levels[level].grids[__beta_i] + (1+pencil+plane);
  double * beta_j = gpu_subdomains[box].levels[level].grids[__beta_j] + (1+pencil+plane);
  double * beta_k = gpu_subdomains[box].levels[level].grids[__beta_k] + (1+pencil+plane);
  double * lambda = gpu_subdomains[box].levels[level].grids[__lambda] + (1+pencil+plane);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0; 
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds){
      // centr of Gershgorin disc is the diagonal element...
      double    Aii = a*alpha[ijk] - b*h2inv*( -beta_i[ijk]-beta_i[ijk+     1]
                                               -beta_j[ijk]-beta_j[ijk+pencil]
                                               -beta_k[ijk]-beta_k[ijk+ plane] );
      lambda[ijk] = 1.0/Aii; // inverse of the diagonal Aii
    }
    ijk+=plane;
  } // for k
} // rebuild_lambda kernel


//==============================================================================================================================================================
__global__ void __initialize_exact(subdomain_type * gpu_subdomains, int level, double hLevel, double a, double b){
  double NPi = 2.0*M_PI;
  double Bmin =  1.0;
  double Bmax = 10.0;
  double c2 = (Bmax-Bmin)/2;
  double c1 = (Bmax+Bmin)/2;
  double c3=10.0; // how sharply (B)eta transitions
  double c4 = -5.0/0.25;

  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int   subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int          pencil = gpu_subdomains[box].levels[level].pencil;
  int           plane = gpu_subdomains[box].levels[level].plane;
  int           low_i = gpu_subdomains[box].levels[level].low.i;
  int           low_j = gpu_subdomains[box].levels[level].low.j;
  int           low_k = gpu_subdomains[box].levels[level].low.k;
  double * grid_u     = gpu_subdomains[box].levels[level].grids[__u_exact] + (plane+pencil+1);
  double * grid_f     = gpu_subdomains[box].levels[level].grids[__f      ] + (plane+pencil+1);
  double * grid_alpha = gpu_subdomains[box].levels[level].grids[__alpha  ] + (plane+pencil+1);
  double * grid_beta  = gpu_subdomains[box].levels[level].grids[__beta   ] + (plane+pencil+1);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  k = 0;
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
      double x = hLevel*((double)(i+low_i)+0.5);
      double y = hLevel*((double)(j+low_j)+0.5);
      double z = hLevel*((double)(k+low_k)+0.5);
      //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
      double r2   = pow((x-0.50),2) +  pow((y-0.50),2) +  pow((z-0.50),2); // distance from center squared
      double r2x  = 2.0*(x-0.50);
      double r2y  = 2.0*(y-0.50);
      double r2z  = 2.0*(z-0.50);
      double r2xx = 2.0;
      double r2yy = 2.0;
      double r2zz = 2.0;
      double r    = pow(r2,0.5);
      double rx   = 0.5*r2x*pow(r2,-0.5);
      double ry   = 0.5*r2y*pow(r2,-0.5);
      double rz   = 0.5*r2z*pow(r2,-0.5);
      double rxx  = 0.5*r2xx*pow(r2,-0.5) - 0.25*r2x*r2x*pow(r2,-1.5);
      double ryy  = 0.5*r2yy*pow(r2,-0.5) - 0.25*r2y*r2y*pow(r2,-1.5);
      double rzz  = 0.5*r2zz*pow(r2,-0.5) - 0.25*r2z*r2z*pow(r2,-1.5);
      //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
      double A  = 1.0;
      //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
      #if 1
      double B  =           c1+c2*tanh( c3*(r-0.25) );
      double Bx = c2*c3*rx*(1-pow(tanh( c3*(r-0.25) ),2));
      double By = c2*c3*ry*(1-pow(tanh( c3*(r-0.25) ),2));
      double Bz = c2*c3*rz*(1-pow(tanh( c3*(r-0.25) ),2));
      #else
      double B  = 1.0;
      double Bx = 0.0;
      double By = 0.0;
      double Bz = 0.0;
      #endif
      //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
      #if 1
      double u   =                exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*sin(NPi*z);
      double ux  = c4*r2x*u + NPi*exp(c4*r2)*cos(NPi*x)*sin(NPi*y)*sin(NPi*z);
      double uy  = c4*r2y*u + NPi*exp(c4*r2)*sin(NPi*x)*cos(NPi*y)*sin(NPi*z);
      double uz  = c4*r2z*u + NPi*exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*cos(NPi*z);
      double uxx = c4*r2xx*u + c4*r2x*ux + c4*r2x*NPi*exp(c4*r2)*cos(NPi*x)*sin(NPi*y)*sin(NPi*z) - NPi*NPi*exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*sin(NPi*z);
      double uyy = c4*r2yy*u + c4*r2y*uy + c4*r2y*NPi*exp(c4*r2)*sin(NPi*x)*cos(NPi*y)*sin(NPi*z) - NPi*NPi*exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*sin(NPi*z);
      double uzz = c4*r2zz*u + c4*r2z*uz + c4*r2z*NPi*exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*cos(NPi*z) - NPi*NPi*exp(c4*r2)*sin(NPi*x)*sin(NPi*y)*sin(NPi*z);
      double f = a*A*u - b*( (Bx*ux + By*uy + Bz*uz)  +  B*(uxx + uyy + uzz) );
      #else
      // should be continuous in u, u', and u''
      // v(w) = w^4 - 2w^3 + w^2
      // u(x,y,z) = v(x)v(y)v(z)
      double X   =  1.0*pow(x,4) -  2.0*pow(x,3) + 1.0*pow(x,2);
      double Y   =  1.0*pow(y,4) -  2.0*pow(y,3) + 1.0*pow(y,2);
      double Z   =  1.0*pow(z,4) -  2.0*pow(z,3) + 1.0*pow(z,2);
      double Xx  =  4.0*pow(x,3) -  6.0*pow(x,2) + 2.0*x;
      double Yy  =  4.0*pow(y,3) -  6.0*pow(y,2) + 2.0*y;
      double Zz  =  4.0*pow(z,3) -  6.0*pow(z,2) + 2.0*z;
      double Xxx = 12.0*pow(x,2) - 12.0*x        + 2.0;
      double Yyy = 12.0*pow(y,2) - 12.0*y        + 2.0;
      double Zzz = 12.0*pow(z,2) - 12.0*z        + 2.0;
      double u   = X*Y*Z;
      double ux  = Xx*Y*Z;
      double uy  = X*Yy*Z;
      double uz  = X*Y*Zz;
      double uxx = Xxx*Y*Z;
      double uyy = X*Yyy*Z;
      double uzz = X*Y*Zzz;
      double f = a*A*u - b*( (Bx*ux + By*uy + Bz*uz)  +  B*(uxx + uyy + uzz) );
      #endif
      //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
      if(withinBounds){
        grid_alpha[ijk] = A;
        grid_beta[ijk]  = B;
        grid_u[ijk]     = u;
        grid_f[ijk]     = f;
      }
      ijk+=plane;
  } // for k
} // initialize exact on gpu kernel


//=============================================================================================================================================================
__global__ void __project_cell_to_face(subdomain_type * gpu_subdomains, int level, int id_cell, int id_face, int dir){
  int box = blockIdx.z;  // CUDA 4 !!!
  int i,j,k; // (0,0,0) = first non ghost zone element
  i = TBDIMX*blockIdx.x + threadIdx.x;
  j = TBDIMY*blockIdx.y + threadIdx.y;

  int subdomain_dim = gpu_subdomains[box].levels[level].dim.i;
  int pencil = gpu_subdomains[box].levels[level].pencil;
  int  plane = gpu_subdomains[box].levels[level].plane;

  double * grid_cell = gpu_subdomains[box].levels[level].grids[id_cell] + (1+pencil+plane);
  double * grid_face = gpu_subdomains[box].levels[level].grids[id_face] + (1+pencil+plane);
  // when the subdomain is coarsened to smaller than a thread block, certain threads won't commit their result -------------------------------------------------
                  int withinBounds = 1;
  if(i>=subdomain_dim)withinBounds = 0;
  if(j>=subdomain_dim)withinBounds = 0;

  int stride;
  switch(dir){
    case 0: stride =      1;break;//i-direction
    case 1: stride = pencil;break;//j-direction
    case 2: stride =  plane;break;//k-direction
  }

  k = 0;
  int ijk = k*plane + j*pencil + i;

  for(k=0;k<subdomain_dim;k++){
    if(withinBounds){
      grid_face[ijk] = 0.5*(grid_cell[ijk-stride] + grid_cell[ijk]); // simple linear interpolation
    }
    ijk+=plane;
  } // for k
} // projection of cell-centered coefficients to face-centered 


//==============================================================================================================================================================
// wrappers...
//==============================================================================================================================================================
extern "C" void zero_grid(domain_type *domain, int level, int grid_id){
  // zeros the grid and the ghost zones
  enqueueEvent(cudaEvent_blas1);
  int box;
  for(box=0;box<domain->numsubdomains;box++){
    hipMemsetAsync(domain->subdomains[box].levels[level].grids[grid_id], 0, domain->subdomains[0].levels[level].volume*sizeof(double),0);
  }
  enqueueEvent(cudaEvent_blas1);
}

extern "C" void initialize_grid_to_scalar(domain_type *domain, int level, int grid_id, double h, double scalar){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __initialize_grid_to_scalar<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,grid_id,h,scalar,level);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void scale_grid(domain_type *domain, int level, int id_c, double scale_a, int id_a){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __scale_grid<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_c,scale_a,id_a,level);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void mul_grids(domain_type *domain, int level, int id_c, double scale, int id_a, int id_b){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __mul_grids<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_c,scale,id_a,id_b,level);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void shift_grid(domain_type *domain, int level, int id_c, int id_a, double scale_a){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __shift_grid<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_c,id_a,scale_a,level);
  enqueueEvent(cudaEvent_blas1);
}

extern "C" void project_cell_to_face(domain_type *domain, int level, int id_cell, int id_face, int dir){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __project_cell_to_face<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,level,id_cell,id_face,dir);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void add_grids(domain_type *domain, int level, int id_c, double scale_a, int id_a, double scale_b, int id_b){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __add_grids<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_c,scale_a,id_a,scale_b,id_b,level);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void rebuild_lambda(domain_type *domain, int level, double a, double b, double hLevel){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __rebuild_lambda<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,a,b,hLevel,level);
  enqueueEvent(cudaEvent_blas1);
}


extern "C" void grid_to_surface_buffers(domain_type *domain, int level, int grid_id){
  enqueueEvent(cudaEvent_s2buf);
  // each thread block copies all minimum stride points for all faces for a given least unit stride offset of one box
  dim3 dimBlock(domain->subdomains[0].levels[level].dim.i,1,1);                    // a stanza of dim points
  dim3 dimGrid(1,domain->subdomains[0].levels[level].dim.i,domain->numsubdomains); // all offsets x all boxes
  __grid_to_surface_buffers<<<dimGrid, dimBlock>>>(domain->gpu_subdomains, grid_id,level);
  enqueueEvent(cudaEvent_s2buf);
}


extern "C" void surface_buffers_to_ghost_buffers(domain_type *domain, int level, int grid_id){
  enqueueEvent(cudaEvent_bufcopy);
  // each thread block copies dim.i points for all faces for a given offset within a face of one box
  dim3 dimBlock(domain->subdomains[0].levels[level].dim.i,1,1);                    // a stanza of dim points
  dim3 dimGrid(1,domain->subdomains[0].levels[level].dim.i,domain->numsubdomains); // all offsets x all boxes
  __surface_buffers_to_ghost_buffers<<<dimGrid, dimBlock>>>(domain->gpu_subdomains, grid_id,level,domain->rank);
  enqueueEvent(cudaEvent_bufcopy);
}

#ifdef __MPI
extern "C" void surface_buffers_to_send_buffer(domain_type *domain, int level, int grid_id){
  enqueueEvent(cudaEvent_pack);
  // each thread block copies dim.i points for all faces for a given offset within a face of one box
  dim3 dimBlock(domain->subdomains[0].levels[level].dim.i,1,1);                    // a stanza of dim points
  dim3 dimGrid(1,domain->subdomains[0].levels[level].dim.i,domain->numsubdomains); // all offsets x all boxes
  __surface_buffers_to_send_buffer<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,domain->gpu_pointers_to_gpu_send_buffer,domain->gpu_pointers_to_gpu_recv_buffer,grid_id,level,domain->rank);
  enqueueEvent(cudaEvent_pack);
}
extern "C" void recv_buffer_to_ghost_buffers(domain_type *domain, int level, int grid_id){
  enqueueEvent(cudaEvent_unpack);
  // each thread block copies dim.i points for all faces for a given offset within a face of one box
  dim3 dimBlock(domain->subdomains[0].levels[level].dim.i,1,1);                    // a stanza of dim points
  dim3 dimGrid(1,domain->subdomains[0].levels[level].dim.i,domain->numsubdomains); // all offsets x all boxes
  __recv_buffer_to_ghost_buffers<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,domain->gpu_pointers_to_gpu_send_buffer,domain->gpu_pointers_to_gpu_recv_buffer,grid_id,level,domain->rank);
  enqueueEvent(cudaEvent_unpack);
}
#endif

extern "C" void ghost_buffers_to_grid(domain_type *domain, int level, int grid_id){
  enqueueEvent(cudaEvent_buf2g);
  // each thread block copies all minimum stride points for all faces for a given least unit stride offset of one box
  dim3 dimBlock(domain->subdomains[0].levels[level].dim.i,1,1);                    // a stanza of dim points
  dim3 dimGrid(1,domain->subdomains[0].levels[level].dim.i,domain->numsubdomains); // all offsets x all boxes
  __ghost_buffers_to_grid<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,grid_id,level);
  enqueueEvent(cudaEvent_buf2g);
}

extern "C" void smooth(domain_type *domain, int level, int phi_id, int rhs_id, double a, double b, double hLevel, int s){
  enqueueEvent(cudaEvent_smooth);
  #ifdef VL
  int IJStride = (VL - 2*(domain->subdomains[0].levels[level].dim.i+2)) & ~0x0F; // i.e. VL minus halos rounded down to a multiple of 16
  if(IJStride==0){printf("Error, IJStride==0 for VL=%d, dim=%d\n",VL,domain->subdomains[0].levels[level].dim.i);exit(0);}
  int NumVectors = ( (domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j)+IJStride+16-1)/IJStride; // i.e. size of a plane less the first/last halo + 16 for rounding
  // FIX, round VLSat up to a multiple of 16
  int VLSat = VL;
  if(VLSat>(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16)
     VLSat=(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16;
  dim3 dimBlock(VLSat,1);
  dim3 dimGrid(NumVectors,1,domain->numsubdomains);
  #else
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1 
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  #endif
  __smooth_once_GSRB<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,phi_id,rhs_id,a,b,hLevel,s,level);
  enqueueEvent(cudaEvent_smooth);
}

extern "C" void restriction(domain_type *domain, int level, int coarse_id, int fine_id){
  enqueueEvent(cudaEvent_restriction);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __restriction<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,fine_id,coarse_id,level); // fine_id@level -> coarse_id@level+1
  enqueueEvent(cudaEvent_restriction);
}

extern "C" void restriction_betas(domain_type *domain, int level){
  enqueueEvent(cudaEvent_restriction);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __restriction_betas<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,level); // update betas at level+1
  enqueueEvent(cudaEvent_restriction);
}

extern "C" void residual(domain_type * domain, int level, int res_id, int phi_id, int rhs_id, double a, double b, double hLevel){
  enqueueEvent(cudaEvent_residual);
  #ifdef VL   
  int IJStride = (VL - 2*(domain->subdomains[0].levels[level].dim.i+2)) & ~0x0F; // i.e. VL minus halos rounded down to a multiple of 16
  if(IJStride==0){printf("Error, IJStride==0 for VL=%d, dim=%d\n",VL,domain->subdomains[0].levels[level].dim.i);exit(0);}
  int NumVectors = ( (domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j)+IJStride+16-1)/IJStride; // i.e. size of a plane less the first/last halo + 16 for rounding
  // FIX, round VLSat up to a multiple of 16
  int VLSat = VL;
  if(VLSat>(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16)
     VLSat=(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16;
  dim3 dimBlock(VLSat,1);
  dim3 dimGrid(NumVectors,1,domain->numsubdomains);
  #else
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  #endif
  __residual<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,res_id,phi_id,rhs_id,a,b,hLevel,level);
  enqueueEvent(cudaEvent_residual);
}

extern "C" void apply_op(domain_type * domain, int level, int     Ax_id, int      x_id, double a, double b, double hLevel){
  enqueueEvent(cudaEvent_apply_op);
  #ifdef VL   
  int IJStride = (VL - 2*(domain->subdomains[0].levels[level].dim.i+2)) & ~0x0F; // i.e. VL minus halos rounded down to a multiple of 16
  if(IJStride==0){printf("Error, IJStride==0 for VL=%d, dim=%d\n",VL,domain->subdomains[0].levels[level].dim.i);exit(0);}
  int NumVectors = ( (domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j)+IJStride+16-1)/IJStride; // i.e. size of a plane less the first/last halo + 16 for rounding
  // FIX, round VLSat up to a multiple of 16
  int VLSat = VL;
  if(VLSat>(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16)
     VLSat=(domain->subdomains[0].levels[level].dim.i+2)*(domain->subdomains[0].levels[level].dim.j+2)+16;
  dim3 dimBlock(VLSat,1);
  dim3 dimGrid(NumVectors,1,domain->numsubdomains);
  #else
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  #endif
  __apply_op<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,Ax_id,x_id,a,b,hLevel,level);
  enqueueEvent(cudaEvent_apply_op);
}


extern "C" void interpolation(domain_type * domain, int level, int fine_id, int coarse_id){ // interpolate from level+1 to level
  enqueueEvent(cudaEvent_interpolation);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __interpolation<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,coarse_id,fine_id,level); // interpolate from level+1 onto level
  enqueueEvent(cudaEvent_interpolation);
}


extern "C" void norm_on_gpu(domain_type * domain, int level, int grid_id, double *gpu_norm){ // norm returned in *gpu_norm.  However, must be hipMemcpy'd to host
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1 
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __norm<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,grid_id,gpu_norm,level);
  enqueueEvent(cudaEvent_blas1);
}

extern "C" void dot_on_gpu(domain_type * domain, int level, int id_a, int id_b, double *gpu_dot){ // dot returned in *gpu_dot.  However, must be hipMemcpy'd to host
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1 
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __dot<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_a,id_b,gpu_dot,level);
  enqueueEvent(cudaEvent_blas1);
}

extern "C" void sum_on_gpu(domain_type * domain, int level, int id_a, double *gpu_sum){ // sum returned in *gpu_sum.  However, must be hipMemcpy'd to host
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1 
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __sum<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,id_a,gpu_sum,level);
  enqueueEvent(cudaEvent_blas1);
}

//=======================================================================================================================================================================
extern "C" void initialize_exact_on_gpu(domain_type * domain, int level, double hLevel, double a, double b){
  enqueueEvent(cudaEvent_blas1);
  int GDIMX = (domain->subdomains[0].levels[level].dim.i+TBDIMX-1)/TBDIMX; // i.e. what happens when TBDIMX==32 on 4^3, GDIMX should always be >=1 
  int GDIMY = (domain->subdomains[0].levels[level].dim.j+TBDIMY-1)/TBDIMY; // i.e. what happens when TBDIMY== 8 on 4^3, GDIMY should always be >=1
  dim3 dimBlock(TBDIMX,TBDIMY);
  dim3 dimGrid(GDIMX,GDIMY,domain->numsubdomains);
  __initialize_exact<<<dimGrid, dimBlock>>>(domain->gpu_subdomains,level,hLevel,a,b);
  enqueueEvent(cudaEvent_blas1);
}
//=======================================================================================================================================================================
extern "C" void ConfigureGPU(){
  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);  // K20x shared memory banking optimized for 64-bit accesses
#ifdef __PREFER_SHARED
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(__smooth_once_GSRB), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(        __residual), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(        __apply_op), hipFuncCachePreferShared);
#else
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(__smooth_once_GSRB), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(        __residual), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(        __apply_op), hipFuncCachePreferL1);
#endif
}
